#include "hip/hip_runtime.h"
#pragma once

#include "hip/hip_runtime.h"
#include ""
#include "VmCommon.h"
#include <stdio.h>
#include "zmq.hpp"

#include "tinythread.h"
#include "OutputBuffer.h"

__constant__ void * packet_data;
__constant__ void * filter_mem;
//__constant__ void * integer_mem;
//__constant__ void * bool_mem;

__constant__ unsigned char program[16 * 1024];
__constant__ unsigned int value_lookup[4 * 1024];
__constant__ short record_size;
__constant__ int packet_count;
__constant__ short filter_count;
__constant__ int filter_size;

// _cache_data holds instructions for caching packet data. It contains two adjacent arrays of operation type and unshifted start offset.
// It provides for up to 16 segments (each covering up to 16 consecutive bytes) which need not be directly adjacent -> support 256 bytes per packet header (excluding optional fields)
// Arrays accessed with SEGOP and SEGX cache functions
__constant__ char _cache_data[32]; //[operation * 16 | offset * 16] 
__constant__ unsigned char _layers; //number of layers

//
//
//#pragma region Cache
//
////stores the start offset of each cache chunk
//
//#define CACHE_LANE		(state.x)
//#define CACHE_SEGOP(id)	(_cache_data[id])		//cache segment load operation (0-4)
//#define CACHE_SEGX(id)	(_cache_data[16 + id])	//unshifted offset in memory
//
//class GpfCache
//{
//	int cache[4];		 //16 byte pachet cache
//	uchar4 filter_cache; //stores upto 32 filter results concurrently and houses cache metadata
//	uchar4 state;		 //cache state variables
//
//	int first_packet;
//
//	int shift;		//only used during caching - extraction is already shifted. 
//					//Some dynamic shift is still possible inside segment, controlled by the dynamic shift
//	
//	__host__ __device__ __forceinline__ int switchByteOrder(uchar4 value)
//	{
//		return (value.x << 24) + (value.y << 16) + (value.z << 8) + value.w;
//	}
//
//	__host__ __device__ void cacheLoad4B(int segment_offset)
//	{
//		int working;
//		//load each packet
//		if (__any(shift))
//		{
//			cache[0] = __ldg(packets + segment_offset + __shfl(shift, 0, 4) + CACHE_LANE);
//			cache[1] = __ldg(packets + segment_offset + recortd_size + __shfl(shift, 1, 4) + CACHE_LANE);
//			cache[2] = __ldg(packets + segment_offset + recortd_size * 2 + __shfl(shift, 2, 4) + CACHE_LANE);
//			cache[3] = __ldg(packets + segment_offset + recortd_size * 3 + __shfl(shift, 3, 4) + CACHE_LANE);
//		}
//		else
//		{
//			cache[0] = __ldg(packets + segment_offset + CACHE_LANE);
//			cache[1] = __ldg(packets + segment_offset + recortd_size + CACHE_LANE);
//			cache[2] = __ldg(packets + segment_offset + recortd_size * 2 + CACHE_LANE);
//			cache[3] = __ldg(packets + segment_offset + recortd_size * 3 + CACHE_LANE);
//		}
//		
//		// 1,0,3,2
//		working = switchByteOrder((uchar4)cache[state.x]);
//		cache[state.x] = __shfl(working, static_cast<int>(state.x), 4);
//
//		// 3,2,1,0		
//		working = switchByteOrder((uchar4)cache[state.y]);
//		cache[state.y] = __shfl(working, static_cast<int>(state.y), 4);
//	
//		// 2,3,1,0
//		working = switchByteOrder((uchar4)cache[state.z]);
//		cache[state.z] = __shfl(working, static_cast<int>(state.z), 4);
//	}
//	
//public:
//	__host__ __device__ GpfCache()
//	{
//		//use shift as a temporary register
//		shift = threadIdx.x & 0x3;		//integer, as shift needs to be shuffled often
//
//		filter_cache.x = 0;							//filter results (32 1-bit boolean values)
//		filter_cache.y = threadIdx.x & 31;			//thread lane in warp
//		filter_cache.z = 0;							//filter segments processed (each corresponds to 8 consecutive filters
//		filter_cache.w = 3 - (filter_cache.y >> 3);	//subwarp filter transform
//
//		state.w = shift;				//thread lane
//		state.x = (5 - shift) & 0x3;	//first transform
//		state.y = 3 - shift;			//second transform
//		state.z = (2 + shift) & 0x3;	//third transform
//		shift = 0;
//	}
//
//	__host__ __device__ __forceinline__ void	resetShift()	{	shift = 0;	}
//	__host__ __device__ __forceinline__ int		getShift()		{	return shift;	}
//
//	__host__ __device__ __forceinline__ void	closeSegment(int segment_shift)	{	shift += segment_shift;	} //adjusts the base shift to incorporate shift introduced in the previous protocol
//
//	// arguments - int operation | int chunk_number (0-15)
//	__host__ __device__ void cacheFill(int segment_id)
//	{
//		int packetIdx = (blockDim.x * blockIdx.x + threadIdx.x);
//		switch(CACHE_SEGOP(segment_id))
//		{
//		case 0:
//		case 1:	
//		case 2: 
//		case 3:
//			cache[program[offsets.x]] = switchByteOrder((uchar4)(
//				__ldg(packets + packet_index * record_size + CACHE_SEGX(segment_id) + shift + CACHE_SEGOP(segment_id))
//				));
//			break;
//		case 4:					//	111...1100 -> pIdx - (pIdx % 4)
//			cacheLoad4B(packetIdx & 0xFFFFFFFC * record_size + CACHE_SEGX(segment_id));
//			break;
//		}
//	}
//	
//	//extracts a field from the cache
//	//bit_offset - the bit offset of the field from the beginning of the segment, including any local shift
//	//bit_length - the bit length of the field
//	//returns - the field
//	__host__ __device__ int extractField(const short2& offsets, const int& dynamic_shift)
//	{
//		int offset = (program[offsets.x + 1] + dynamic_shift) & 128;
//		int length = program[offsets.x + 2];
//		//load cache integer, trimming leading bits
//		int out = cache[offset >> 32] & (0xFFFFFFFF >> (offset & 31));
//	
//		//if field contained in one int, trim any trailing bits
//		if (offset & 31 + length < 33) return out >> (32 - offset & 31 - length);
//		//else merge with next chunk
//		else return (out << (offset + length) & 31) + (cache[1 + (offset >> 32)] >> (32 - (offset + length) & 31));
//	}
//	
//	__host__ __device__ void storeFilter(int& bool_reg, short2& offsets)
//	{
//		int working = __ballot((bool_reg >> (31 - program[offsets.x])) & 0x1); //gets the results for one rule in bool_reg using ballot to poll
//
//		//trim to correct byte for threads subwarp
//
//		working = (working >> filter_cache.w) & 0xFF;
//
//		filter_cache.x |= static_cast<unsigned char>(working & (threadIdx.x & 7 == program[offsets.x] & 7) ? 0xFF | 0x0);
//		if  (program[offsets.x]  & 7 == 7)//just wrote 8th rule to filter mem
//		{
//			//currently have 4 sets of results (one for each 8 thread subwarp). Each thread holds 8 results for a single filter.
//			//Each filter is spread over 4 threads in section [0-7][8-15][16-23][24-31]
//			//Filter chars can be combined into a single int and written, using 8 of the 32 threads
//
//			//fist merge [0-7] & [8-15] / [16-23]&[24-31] threads
//			//			
//			/*working = filter_cache.x << (((3 - (filter_cache.y >> 3)) & 0x1) << 3);
//			working |= _shfl_down(working, 8, 16);
//
//			working = working << (((15 - (filter_cache.y >> 2)) & 0x1) << 4);
//			working |= _shfl_down(working, 16, 32);*/
//
//			//	00   01	  02   03	04	 05	  06   07	<-- threadIdx	
//			//[0,0][1,0][2,0][3,0][4,0][5,0][6,0][7,0]	<-- Filter[filter number, offset shift]
//			//
//			//	08	 09	  10   11   12	 13	  14   15
//			//[0,1][1,1][2,1][3,1][4,1][5,1][6,1][7,1]
//			//
//			//	16	 17	  18   19	20	 21	  22   23
//			//[0,2][1,2][2,2][3,2][4,2][5,2][6,2][7,2]
//			//
//			//	24	 25	  26   27	28	 29	  30   31
//			//[0,3][1,3][2,3][3,3][4,3][5,3][6,3][7,3]
//
//			static_cast<unsigned char*>(filter_mem)[filter_size * ((filter_cache.z << 3) + (threadIdx.x & 7)) + (filter_cache.y >> 3)] = filter_cache.x;
//		
//			++filter_cache.z;	//increment base offset
//		}
//	}
//	/*__host__ __device__ void cacheSlide(int* cache, int& x)
//	{
//		int slide_start = program[x + 2];
//		int slide_width = program[x + 3];
//		for (int k = 0; k < slide_width; k++)
//		{
//			cache[k] = cache[slide_start + k];
//		}
//	}*/
//}
//
//#pragma endregion
//
//#pragma region Processor
//
////#define SET_PROTOCOL 0
////#define JUMP 0
////#define JUMP_TRUE 0
////#define JUMP_FALSE 0
//
//#define EXTRACT_FIELD_TMP 0
////#define EXTRACT_FIELD_REG 0
////
////#define STORE_TMP_REG 0
////#define STORE_TMP_GLOBAL 0
////#define STORE_REG_GLOBAL 0
////#define STORE_GLOBAL_REG 0
////#define STORE_GLOBAL_TMP 0
//
//#define STORE_FILTER 0
//
//#define RULE_COMPARISON_REG 0
//#define RULE_COMPARISON_TMP 0
//
//#define STATE_PX	(offsets.x)
//#define STATE_		(offsets.y)
//
//#define ACTIVE_SEGMENT	(_any
//#define PROTO_ID		(proto_state.x)
//#define PROTO_NEXT		(proto_state.y)
//#define PROTO_START		(proto_state.z)
//#define PROTO_LENGTH	(proto_state.w)
//
//class GpfVm
//{
//	//int int_reg[8];
//	//int bool_reg;
//	int bool_reg;
//	short2 offsets;		//stores the offsets of the program pointer and data shift register
//	uchar4 proto_state;	//stores the primary state of the active protocol - id, data span and payload
//	int tmp;			//externally accessible semi coherent working register
//	int working;		//internal non-coherent working register
//	
//	int local_shift;
//	uchar4 control;
//	GpfCache cache;
//
//public:
//
//	__host__ __device__ GpfVm() : bool_reg(0), tmp(0), working(0)
//	{
//		//state.x = 1;	//layer loop
//		//state.y = 1;	//segment loop
//		control = (uchar4)0x01010000;
//		
//		//PROTO_ID = 1; //first (root) protocol
//		//PROTO_NEXT = 0; //Data
//		//PROTO_START = 0; //data always trimmed
//		//PROTO_LENGTH unknown at this point - may be ignored if protocol is statically sized
//		proto_state =  (uchar4)0x01000000;
//	
//	}
//
//	__host__ __device__ void Process()
//	{
//		
//		while (control.x < _layers)
//		{
//			//load cache
//			cache.cacheFill(k);
//
//			//no need to skip cache loads, as the proto_id test ensures that processing terminates when 
//			//protocol goes out of bounds. as this is the same test used to determine if caching should
//			//be performed, processing will always terminate before the test would be of any value.
//
//
//			//process associated segment
//			while (control.y < _segments[control.x])
//			{
//				//warp vote to determine whether to process the segment
//				//if all threads do not match the segment, it can be skipped
//				if (__all(PROTO_NEXT - program[offsets.x]))
//				{
//					//skip over segment 
//					offsets.x += program[offsets.x + 1];
//					continue;
//				}
//				local_shift = 0;
//				++control.y;	//increment early to avoid read after write conflict
//				offsets.x += 2;
//				processSegment();
//			}
//
//			++control.x;		//increment early to avoid read after write conflict
//			if (__any(PROTO_ID) == 0) break; //escape if all warp protocols are null
//		}
//	}
//
//	__host__ __device__ void processSegment()
//	{
//		while (program[offsets.x] != 0xFF)
//		{
//			switch(program[offsets.x])
//			{
//
//				case TERMINATE_HEADER:
//					if (PROTO_ID == program[offsets.x + 1]) 
//					{
//						proto_state.x = 0; //dont write proto_id and proto_next sequentially since they are same register
//						proto_state.y = 0;
//					}
//					offsets.x += 2;
//					break;
//					
//					//Extract
//				case EXTRACT_FIELD_TMP:	//need to add local shift
//					tmp = cache.extractField(offsets, local_shift);
//					offsets.x += 3;
//					break;
//
//					//Rule Comparison
//				case RULE_COMPARISON_REG:
//					ruleComparison(int_reg[program[offsets.x + 5]]);
//					offsets.x += 6;
//					break;
//				case RULE_COMPARISON_TMP:
//					ruleComparison(tmp);
//					offsets.x += 5;
//					break;
//
//				case STORE_FILTER:
//					cache.storeFilter(bool_reg, offsets);
//			}
//		}
//
//				//	//Protocl
//				//case SET_PROTOCOL:
//				//	if (PROTO_ID == program[x]) 
//				//	{
//				//		PROTO_ID = PROTO_NEXT;
//				//		PROTO_NEXT = 0;
//				//	}
//				//	break;
//				//case JUMP:
//				//	break;
//				//case JUMP_FALSE:
//				//	extractBool(bool_reg, x, working);
//				//	if (__all(working)) //!=0
//				//	{
//				//	}
//				//	else //dont jump - process decisional code
//				//	{
//				//	}
//				//	break;
//				//case JUMP_TRUE:
//				//	extractBool(bool_reg, x, working);
//				//	if (__all(working) == 0) 
//				//	{
//				//	}
//				//	else
//				//	{
//				//	}
//				//	break;
//				//case SHIFT_DATA:
//				//	extractBool(bool_reg, x, working);
//				//	if (PROTO_ID == program[x]) 
//				//	{
//				//		protocol _id = PROTO_NEXT;
//				//		PROTO_NEXT = -1;
//				//	}
//				//	break;
//				//case EXTRACT_FIELD_REG:
//				//	int_reg[program[x + 4]] = extractField(cache, x);
//				//	break;
//				//	//Store INT
//				//case STORE_TMP_REG:
//				//	int_reg[program[x + 2]] = tmp;
//				//	break;
//				//case STORE_TMP_GLOBAL:
//				//	integer_mem[program[x + 2] * packet_count + (blockDim.x * blockIdx.x) + threadIdx] = tmp;
//				//	break;
//				//case STORE_REG_GLOBAL:
//				//	integer_mem[program[x + 2] * packet_count + (blockDim.x * blockIdx.x) + threadIdx] = int_reg[program[x + 3]];
//				//	break;
//				//case STORE_GLOBAL_REG:
//				//	int_reg[program[x + 2]] = integer_mem[program[x + 3] * packet_count + (blockDim.x * blockIdx.x) + threadIdx];
//				//	break;
//				//case STORE_GLOBAL_TMP:
//				//	tmp = integer_mem[program[x + 2] * packet_count + (blockDim.x * blockIdx.x) + threadIdx];
//				//	break;
//				//case STORE_FILTER:
//				//	switch(program[x + 2])
//				//	{
//				//	case 0:
//				//	}
//				//	break;
//				//	
//				//	//BOOL Comparison
//				//case BOOL_COMPARISON_REG:
//				//	ruleComparison(int_reg[program[x + 2]], bool_reg, x);
//				//	break;
//				//case BOOL_COMPARISON_TMP:
//				//	ruleComparison(tmp, bool_reg, x);
//				//	break;
//				//}
//	}
//	
//	#define COMPARISON_EQU 0
//	#define COMPARISON_NEQ 1
//	#define COMPARISON_LES 2
//	#define COMPARISON_GRT 3
//	#define COMPARISON_LEQ 4
//	#define COMPARISON_GEQ 5
//
//	//	Compares the value of in to a value in program memory, 
//	__host__ __device__ void RuleComparison(int* field)
//	{
//		working = 0x1 << (31 - program[x + 4]);
//		switch(program[offsets.x + 2])
//		{
//			case COMPARISON_EQU: 
//				bool_reg = (PROTO_ID == program[offsets.x + 1]) && (*field == program[x + 3]) ? bool_reg | working : bool_reg & ~working;
//				break;
//			case COMPARISON_NEQ: 
//				bool_reg = (PROTO_ID == program[offsets.x + 1]) && (*field != program[x + 3]) ? bool_reg | working : bool_reg & ~working;
//				break;
//			case COMPARISON_LES:
//				bool_reg = (PROTO_ID == program[offsets.x + 1]) && (*field < program[x + 3]) ? bool_reg | working : bool_reg & ~working;
//				break;
//			case COMPARISON_GRT: 
//				bool_reg = (PROTO_ID == program[offsets.x + 1]) && (*field > program[x + 3]) ? bool_reg | working : bool_reg & ~working;
//				break;
//			case COMPARISON_LEQ: 
//				bool_reg = (PROTO_ID == program[offsets.x + 1]) && (*field <= program[x + 3]) ? bool_reg | working : bool_reg & ~working;
//				break;
//			case COMPARISON_GEQ:
//				bool_reg = (PROTO_ID == program[offsets.x + 1]) && (*field >= program[x + 3]) ? bool_reg | working : bool_reg & ~working; 
//				break;
//		}
//
//	}
//	//
//	//__device__ inline void extractBool(int& reg, int& x, int& out)
//	//{
//	//	out = (reg >> program[x + 2]) & 0x1; 
//	//}
//
//}
//
//__global__ void GpfProcessor(int stream_no)
//{
//	
//}
//
//#pragma endregion

#pragma region Host

void Process(void * args);

void Setup(FilterOptions& options)
{
	//hipMalloc((void**) &packet_data, options.packet_buffer_size());
	//hipMalloc((void**) &filter_mem, options.filter_memory_size());

}

static void GpfVmLauncher(zmq::context_t * zmq_context, FilterOptions filter_options)
{
	/*ProcessArgs * args = static_cast<ProcessArgs*>(malloc(sizeof(ProcessArgs)));
	args->context = zmq_context;
	args->options = filter_options;
	tthread::thread* proc = new tthread::thread(ProcessFilter, (void*) args);*/
}

void ProcessFilter(void * args)
{
	//ProcessArgs * proc = static_cast<ProcessArgs*>(args);
	//zmq::socket_t buffer(proc->context, ZMQ_PAIR);
	//buffer.connect("inproc://gpfbuffer");

	//OutputBuffer output(proc->context, proc->options);

	//CudaBufferPointer ptr;

	//FilterOptions options = proc->options;
	//Setup(options);	//setup the vm memory regions (once off)
	//
	////create streams
 //   hipStream_t *streams = (hipStream_t*) malloc(options.streams * sizeof(hipStream_t));
 //   for(int i = 0; i < options.streams; i++) {
 //       checkCudaErrors( hipStreamCreate(&(streams[i])) );
 //   }

	//int curr_stream = 0;
	//_int64 packet_count = 0;
	//do
	//{
	//	buffer.recv(&ptr, sizeof(CudaBufferPointer));	//get next full write-combined buffer

	//	if (ptr.size != options.packet_buffer_stream_size()) //last pointer - use default streams
	//	{
	//		hipMemcpyAsync(packet_data, ptr.buffer, ptr.size, hipMemcpyHostToDevice, 0);
	//		GpfProcessor<<<options.blocks(), options.threads>>>(0);
	//		
	//		if (options->filter_results()) hipHostAlloc((void**) &filter_results,  options->filter_memory_size(), hipHostMallocDefault); 
	//		if (options->integer_results()) hipHostAlloc((void**) &integer_results, options->integer_memory_size(), hipHostMallocDefault); 
	//	}
	//	//copy packet buffer to device memory
	//	for (int k = 0; k < options.streams; k++)
	//	{
	//		//copy records to the device - async
	//		hipMemcpyAsync(packet_data + k * options.packets_per_stream(), 
	//						ptr.buffer + k * options.packets_per_stream(), 
	//						options.packet_buffer_stream_size(ptr.size, k), 
	//						hipMemcpyHostToDevice, 
	//						streams[k]);

	//		//process stream contents - async
	//		/*GpfProcessor<<<options.blocks(), options.threads, 0, streams[k]>>>(k);*/
	//	}
	//	char* filter_results;
	//	int* integer_results;

	//	//retreive results - async
	//	if (options->filter_results()) hipHostAlloc((void**) &filter_results, options->filter_memory_size(), hipHostMallocDefault); 
	//	if (options->integer_results()) hipHostAlloc((void**) &integer_results, options->integer_memory_size(), hipHostMallocDefault); 

	//	if (options->filter_results()) 
	//	{
	//		hipMemcpy(filter_results, filter_mem, options->filter_memory_size(), hipMemcpyDeviceToHost, 0); //use defaults stream
	//		output.CopyFilterResults(filter_results, options->filter_memory_size());
	//	}
	//	if (options->integer_results()) 
	//	{
	//		hipMemcpy(integer_results, integer_mem, options->integer_memory_size(), hipMemcpyDeviceToHost, 0); //use defaults stream
	//		output.CopyFilterResults(integer_results, options->integer_memory_size());
	//	}

	//} while (ptr.more);
	////complete

}

#pragma endregion