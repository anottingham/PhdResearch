#include "hip/hip_runtime.h"

#pragma once

#include "hip/hip_runtime.h"
#include ""
#include "VmCommon.h"
#include <stdio.h>
#include "zmq.hpp"

#include "tinythread.h"
#include "OutputBuffer.h"

#include "GpfVm.cuh"

#pragma region Defines

//#define GPF_DEBUG

//#define PERFORMANCE_TESTING
//thread count and streams in VmCommon
#define ITERATIONS				32		//number of iterations before filtering - not user configurable

#define GPF_WARPS				(GPF_THREADS/32)

#define PACKETS_PER_BLOCK		(GPF_THREADS * ITERATIONS)
#define PACKETS_PER_WARP		(PACKETS_PER_BLOCK / GPF_WARPS)

#define PROGRAM_X				offsets.x		//the program counter base offset
#define PROTOCOL_X				offsets.y		//the start byte offset of the current protocol

#define PROGRAM(x)				rule_program[PROGRAM_X + x] //the value in program memory x indeces past the program counter
#define FILTER_PROGRAM(x)		fltr_program[PROGRAM_X + x]

#define PROTO_ID				proto_state.x //Identifies the protocol
#define PROTO_NEXT				proto_state.y	//Identifies the next protocol
#define PROTO_LENGTH			proto_state.z	//Protocol length (Bytes)
#define PROTO_CURR				proto_state.w	//Identifies the active protocol (may or may not be relevant to current thread, but will be relevant to a thread in the warp)
#define PROTO_LEN_BITS			(proto_state.z << 3)


#define WARP_ID					thread_state.x		//threads warp index
#define STREAM					thread_state.y
#define ITERATION				thread_state.z		//current iteration
#define ALIGNMENT				thread_state.w		//byte alignment in integer [0:3]

#define CACHE_LANE				transform.w
#define PACKET_COUNT			_packet_count

#define COMPARISON_EQU 0
#define COMPARISON_NEQ 1
#define COMPARISON_LES 2
#define COMPARISON_GRT 3
#define COMPARISON_LEQ 4
#define COMPARISON_GEQ 5

#define EXPR_READ_TMPMEM	0
#define EXPR_READ_SYSREG	1
#define EXPR_READ_LOOKUP	2

#define EXPR_WRITE_TMPMEM	0
#define EXPR_WRITE_SYSREG	1

#define PRED_WRITE_TMPMEM	0
#define PRED_WRITE_FILTER	1

#define PRED_READ_TMPMEM	0
#define PRED_READ_FILTER	1



//each block has 512 threads - each processing 32 packets per rule - so each rule requires 512 ints to store
//each block writes 512 / 32 = 8  ints per iteration
//each warp writes a single int at the index equivalent to their warp id after the start of the iteration
//#define WORKING_WRITE(rule)			working_mem[_working_per_block * blockIdx.x + rule * GPF_THREADS + ITERATION * (GPF_THREADS >> 5) + WARP_ID]
//#define WORKING_READ(rule)			working_mem[_working_per_block * blockIdx.x + rule * GPF_THREADS + threadIdx.x]


#define WORKING_WRITE(rule)			working_mem[STREAM * _working_per_stream + _working_per_block * blockIdx.x + _working_per_warp * WARP_ID + rule * 32 + ITERATION]
#define WORKING_READ(rule)			working_mem[STREAM * _working_per_stream + _working_per_block * blockIdx.x + _working_per_warp * WARP_ID + rule * 32 + (threadIdx.x & 31)]

#define INTEGER_RW(index)			integer_mem[STREAM * _integer_per_stream + _integer_mem_size * index + packet_index]

//#define FILTER_RW(index)			filter_mem[_filter_mem_size * (_bool_result_count * STREAM  + index) + blockDim.x * blockIdx.x + threadIdx.x]
#define FILTER_RW(index)			filter_mem [STREAM * _filter_per_stream  + _filter_mem_size * index + blockDim.x * blockIdx.x + threadIdx.x]
#pragma endregion

#pragma region Constant Memory

__constant__ int * packet_data;		//memory region containing trimmed packet data
__constant__ int * filter_mem;		//memory region containing 1-bit filter results
__constant__ int * integer_mem;		//memory region containing 32-bit integer results
__constant__ int * working_mem;

__constant__ unsigned char rule_program[14 * 1024];		//the main (rule) program
__constant__ unsigned char fltr_program[2 * 1024];		//1-bit filter evaluation program (post-process)
__constant__ int lookup[255];							//32-bit integer lookup memory
__constant__ int _packet_count;							//number of packets per streamed batch

__constant__ int _record_size;					//the size of a trimmed/padded packet record in ints
__constant__ int _filter_mem_size;				//size of filter mem per result in ints
__constant__ int _integer_mem_size;				//size of integer mem per result in ints

__constant__ int _bool_result_count;			//number of unique boolean results per packet
__constant__ int _int_result_count;				//number of unique integer results per packet

__constant__ int _layer_count;					//number of layers in the rule program
__constant__ int _rules_per_packet;				//number of rules per packet in working memory

__constant__ char _root_proto;					//the root protocol (ethernet, 802.11 etc)
__constant__ char _start_offset;				//the number of bits between the start of the packet and the first referenced field (used to account for trimming of the first protocol)

__constant__ int _working_per_block;
__constant__ int _working_per_warp;

__constant__ int _working_per_stream;
__constant__ int _filter_per_stream;
__constant__ int _integer_per_stream;

#pragma endregion

#pragma region Device Methods

__device__ GpfVm::GpfVm(int stream)
{
	////breifly use shared memory to broadcast the working memory pointer to all threads in block
	//__shared__ int * tmp;
	//	
	////use shift as a temporary register

	////use shared memory to broadcast working memory pointer
	//if (threadIdx.x == 0)
	//{
	//	//_working_per_block = _rules_per_packet * blockDim.x (GPF_THREADS)
	//	// working memory organsied as a set of contiguous rule results (512 ints -> 16384 packets). iteration is index for each rule (32 total)
	//	tmp = (int*) malloc(_working_per_block);
	//	if (tmp == nullptr) printf("null working in block %d for some reason..\n", blockIdx.x);
	//}
	//__syncthreads();
	
	//sanitize
	

	WARP_ID = threadIdx.x >> 5;			//warp number
	CACHE_LANE = threadIdx.x & 3;			//thread lane in group
	ITERATION = 0;							//iterations			
	STREAM = stream;
	
	//if (blockIdx.x  == 30 && threadIdx.x == 192)
	//{
	//	printf("Warp ID = %d\nPPW = %d\nProduct = ",WARP_ID,PACKETS_PER_WARP,WARP_ID * PACKETS_PER_WARP);
	//}
	packet_index = blockIdx.x * PACKETS_PER_BLOCK + WARP_ID * PACKETS_PER_WARP + (threadIdx.x & 31);

	transform.x = (5 - CACHE_LANE) & 0x3;	//first transform
	transform.y = 3 - CACHE_LANE;			//second transform
	transform.z = (2 + CACHE_LANE) & 0x3;	//third transform	

	
	//0 each iteration of WPP rules with a single command
	for (int k = 0; k < _rules_per_packet; k++)
	{
		//ensure thread only handles its own local memory - warps are not guarenteed to exeute in unison
		//early warp could start writing rules before later warp has opertunity to zero memory
		WORKING_READ(k) = 0;
	}//0 each iteration of WPP rules with a single command
}
	
#pragma region Processor

__device__ void GpfVm::Gather()
{
#pragma unroll 32
	for (int k = 0; k < 32; k++)
	{
	/*	if (blockIdx.x  == 27 && threadIdx.x == 0)
		{
			printf("Iteration = %d\n",ITERATION);
		}*/
		
		PROGRAM_X = 0;	//reset program
		PROTOCOL_X = 0;

		PROTO_ID = 0;
		PROTO_NEXT =  packet_index < PACKET_COUNT ? _root_proto : 0; //physical layer specified in capture header
		PROTO_LENGTH = 0;
		PROTO_CURR = 0;
		ALIGNMENT = 0;

		//sanitise integer mem
		for (int j = 0; j < _int_result_count; j++)
		{
			INTEGER_RW(j) = 0;
		}
		
		if (__ballot(PROTO_NEXT > 0) == 0)
		{
			break; //break out if all protocols are out of range
		}

		ProcessPacket();
		
		packet_index += 32;	//blockDim.x packets per iteration
		++ITERATION; 
	}
}

__device__ void GpfVm::Filter()
{
	//process filters

	if (GPF_THREADS * blockIdx.x + threadIdx.x < (PACKET_COUNT >> 5) + (PACKET_COUNT & 31 == 0 ? 0 : 1)) //in range (32 packets per thread)
	{
		/*if (threadIdx.x == 437 && blockIdx.x == 7)
		{
			printf("something here");
		}*/

		//__syncthreads();
		PROGRAM_X = 4; //adjustment due to constant cache problem. Remove once resolved.
		uchar4 loop_counter = make_uchar4(0,0,0,0);
		uchar4 loop_boundry;

	/*	if (blockIdx.x == 0 && threadIdx.x == 0) 
		{
			printf("Filter Memory:\n\t");
			for (int k = 0; k < 16; k++) printf(" %d",  FILTER_PROGRAM(k));
			printf("\n");
		}*/

		while (loop_counter.x < _bool_result_count)
		{
			++loop_counter.x;
			/*if (blockIdx.x == 0 && threadIdx.x == 0) 
			{
				printf("Filter Memory:\n\t");
				for (int k = 0; k < 8; k++) printf(" %d",  FILTER_PROGRAM(k));
				printf("\n");
			}*/

			loop_counter.y = 0;
			loop_boundry.y = FILTER_PROGRAM(0);	//transaction count
			++PROGRAM_X;

			while (loop_counter.y < loop_boundry.y)
			{
				loop_counter.z = 0;
				loop_boundry.z = FILTER_PROGRAM(0);	//or count
				++PROGRAM_X;
				int ans = 0;
				
				while (loop_counter.z < loop_boundry.z)
				{
					loop_counter.w = 0;
					loop_boundry.w = FILTER_PROGRAM(0);	//and count
					++PROGRAM_X;
					int and = 0xFFFFFFFF;

					while (loop_counter.w < loop_boundry.w)
					{
						
						int value = FILTER_PROGRAM(0) == 1
							?	~WORKING_READ(FILTER_PROGRAM(1))
							:	WORKING_READ(FILTER_PROGRAM(1))
							;

						/*if (blockIdx.x == 0 && threadIdx.x < 16) 
						{
							printf("Packet Group %d:\n\tTransaction Count = %d\n\tOr Count = %d\n\tAnd Count = %d\n\tValue = %x\n\tWorking = %d\n", GPF_THREADS * blockIdx.x + threadIdx.x, loop_boundry.y, loop_boundry.z, loop_boundry.w, value, WORKING_READ(FILTER_PROGRAM(1)));
						}*/
						and &= value;
						/*if (blockIdx.x == 0 && threadIdx.x < 4) 
						{
							printf("Packet Group %d:\n\tAnd Value = %x\n\tValue = %x\n", GPF_THREADS * blockIdx.x + threadIdx.x, and, value);
						}*/
						++loop_counter.w;
						PROGRAM_X += 2;
					}
					
					++loop_counter.z;
					ans |= and;
				}
				++loop_counter.y;
				
				/*if (blockIdx.x == 0 && threadIdx.x < 4) 
				{
					printf("Packet Group %d:\n\tTransaction Count = %d\tProgram_x = %d\n", GPF_THREADS * blockIdx.x + threadIdx.x, loop_boundry.y, PROGRAM_X);
				}*/

				if (FILTER_PROGRAM(0) == PRED_WRITE_TMPMEM)
				{
					WORKING_READ(FILTER_PROGRAM(1)) = ans;
				}
				//case: PRED_WRITE_FILTER:
				else
				{	
					/*if (ans == 0xFFFFFFFF && FILTER_PROGRAM(1) == 0)
					{
						printf("something here");
					}*/
					FILTER_RW(FILTER_PROGRAM(1)) = __byte_perm(__brev(ans), 0, 0x0123);
					/*if (blockIdx.x == 22 && threadIdx.x < 16) 
					{
						printf("Packet Group %d:\n\tAns = %x\n\Filter[Ans] = %x\n", GPF_THREADS * blockIdx.x + threadIdx.x, ans, FILTER_RW(FILTER_PROGRAM(1)));
					}*/
				}
				PROGRAM_X += 2;
			}
		}
	}
		
	//clear working memory after all threads have completed
	/*__syncthreads();

	if (threadIdx.x == 0)
	{
		free(working_mem);
	}*/
}

__device__ void GpfVm::ProcessPacket()
{
	//if (this == nullptr || ITERATION > 31)
	//{
	//	printf("SOmething is wrong...");
	//	return;
	//}

	uchar4 loop_control = make_uchar4(0,0,0,0);

	while (loop_control.x < _layer_count)
	{
		loop_control.y = 0;
		loop_control.z = PROGRAM(0);

		PROGRAM_X += 1;
		//test protocols for relevance / perform protocol setup if relevant
		while (loop_control.y++ < loop_control.z)
		{
			if (PROGRAM(0) == PROTO_NEXT)
			{
				PROTO_ID = PROTO_NEXT;
				PROTO_NEXT = 0;
				PROTOCOL_X += PROTO_LENGTH;
				PROTO_LENGTH = PROGRAM(1);
			}
			PROGRAM_X += 2;
		}
		
		
		if (__all(PROTO_NEXT) != 0) {
			++loop_control.x;		//increment early to avoid read after write conflict
			PROGRAM_X += PROGRAM(0);
			continue; //all packets out of headers - break out of layer loop
		}
		

		
		//if (blockIdx.x == 0 && threadIdx.x < 4) 
		//{
		//	printf("Packet %d Layer %d :\n\tProto = %d\n\tLength = %d\n\tX = %d\n\tProgram X = %d\n", packet_index, loop_control.x, PROTO_ID, PROTO_LENGTH, PROTOCOL_X, PROGRAM_X);
		//}
			
		//segemt count
		uchar4 inner_loop;
		inner_loop.x = 0;
		inner_loop.y = PROGRAM(1);
		PROGRAM_X += 2;

	/*	if (blockIdx.x == 0 && threadIdx.x < 4) 
		{
			printf("Packet %d Layer %d :\n\tSegment Count = %d\n\tProgram_x = %d\n", packet_index, loop_control.x, PROGRAM(0),  PROGRAM_X);
		}*/

		//no need to skip cache loads, as the proto_id test ensures that processing terminates when 
		//protocol goes out of bounds. as this is the same test used to determine if caching should
		//be performed, processing will always terminate before the test would be of any value.


		//process associated segment
		while (inner_loop.x < inner_loop.y)
		{
			++inner_loop.x;	//increment early to avoid read after write conflict
			
			inner_loop.z = 0;
			
		/*	if (blockIdx.x == 0 && threadIdx.x < 4) 
			{
				printf("Packet %d Layer %d :\n\tSet Count = %d\n\tCache Load = %d\n", packet_index, loop_control.x, PROGRAM(0), PROGRAM(1));
			}*/

			//fill cache for the segment
			CacheLoad(packet_data + (STREAM * PACKET_COUNT + packet_index & 0xFFFFFFFC) * _record_size + CACHE_LANE);
			
			inner_loop.w = PROGRAM(0);
			++PROGRAM_X;


			//no cooperation required after this point

			while(inner_loop.z < inner_loop.w)
			{
				++inner_loop.z;
				
				/*if (blockIdx.x == 0 && threadIdx.x < 4) 
				{
					printf("Packet %d Layer %d :\n\tProto = %d\n\tProgram = %d\n\tNext = %d\n", packet_index, loop_control.x, PROTO_ID, PROGRAM(0), PROGRAM(1));
				}*/
				
				
				/*if (blockIdx.x == 0 && threadIdx.x < 4 && ITERATION < 4) 
				{
					printf("Packet %d :\n\tProgram_x = %d\n", packet_index,  PROGRAM_X);
				}*/
				//warp vote to determine whether to process the segment
				//if all threads fail to match the segment, it can be skipped
				if (__all(PROTO_ID - PROGRAM(0)))
				{
					/*if (blockIdx.x == 0 && threadIdx.x < 4 && ITERATION < 4) 
					{
						printf("Packet %d Skipped:\n\tAll = \tProgram_x = %d\n\tStart = %d\n", packet_index, __all(PROTO_ID - PROGRAM(0)), PROGRAM_X);
					}*/
					//skip over segment 
					PROGRAM_X += PROGRAM(1);
					continue;
				}

				PROTO_CURR = PROGRAM(0);
				PROGRAM_X += 2;
				/*if (blockIdx.x == 0 && threadIdx.x < 4 && ITERATION < 4) 
				{
					printf("Packet %d :\n\tProgram_x = %d\n", packet_index,  PROGRAM_X);
				}*/
				ProcessSet();
			}
		}

		++loop_control.x;		//increment early to avoid read after write conflict
		if (__any(PROTO_NEXT) == 0) break; //escape if all warp protocols are null
		PROTO_ID = 0;
	}
}
	


//processes a set for a specifc protocol
__device__ void GpfVm::ProcessSet()
{
	uchar4 set_mem;
	set_mem.x = 0;				//current set
	set_mem.y = PROGRAM(0);		//set count

	unsigned int value;

	++PROGRAM_X;

	/*if (blockIdx.x == 0 && threadIdx.x < 4 && ITERATION < 4) 
	{
		printf("Packet %d :\n\tProgram_x = %d\n", packet_index,  PROGRAM_X);
	}*/
	//each set contains one or more fields, which may in tern contain one or more filters
	//iterate through each field, extracting it and filtering it
	while (set_mem.x < set_mem.y)
	{
		//iterator not used again so can increment early
		++set_mem.x;
		value = ExtractField();

		//store field value if it is needed as an integer result
		if (PROGRAM(0) != 0xFF)
		{
			INTEGER_RW(PROGRAM(0)) = PROTO_ID == PROTO_CURR ? value : 0;
		}

		set_mem.z = 0;
		set_mem.w = PROGRAM(1);
		PROGRAM_X += 2;
			
		while (set_mem.z < set_mem.w)
		{
			++set_mem.z;
			bool ans = false;
			switch (PROGRAM(0))
			{
			case COMPARISON_EQU:
				ans = PROTO_ID == PROTO_CURR && value == lookup[PROGRAM(1)];
				break;
			case COMPARISON_NEQ:
				ans = PROTO_ID == PROTO_CURR && value != lookup[PROGRAM(1)];
				break;
			case COMPARISON_LES:
				ans = PROTO_ID == PROTO_CURR && value < lookup[PROGRAM(1)];
				break;
			case COMPARISON_GRT:
				ans = PROTO_ID == PROTO_CURR && value > lookup[PROGRAM(1)];
				break;
			case COMPARISON_LEQ:
				ans = PROTO_ID == PROTO_CURR && value <= lookup[PROGRAM(1)];
				break;
			case COMPARISON_GEQ:
				ans = PROTO_ID == PROTO_CURR && value >= lookup[PROGRAM(1)];
				break;
			}

			/*if (blockIdx.x == 0 && threadIdx.x < 4 && ITERATION < 4) 
			{
				if (ans) printf("Packet %d :\n\t Value = %d\n\tCompValue = %d\n\tAns = true\n", packet_index, value, lookup[PROGRAM(1)]);
				else printf("Packet %d :\n\t Value = %d\n\tCompValue = %d\n\tAns = false\n", packet_index, value, lookup[PROGRAM(1)]);
			}*/

			if (PROGRAM(2) > 0 && ans) PROTO_NEXT = PROGRAM(2);
			if (PROGRAM(3) != 0xFF)
			{
				
				int tmp = __ballot(ans ? 1 : 0);
				if ((threadIdx.x & 31) == 0)
				{			
					WORKING_WRITE(PROGRAM(3)) |= tmp;

					/*if (blockIdx.x == 0 && threadIdx.x < 4 && ITERATION < 16) 
					{
						printf("Packet %d + 32 :\n\t Working Store = %x\n\tWorking Index = %d\n", packet_index, WORKING_READ(PROGRAM(3)), PROGRAM(3));
					}*/
				}
			}
			PROGRAM_X += 4;
		}


		//prepare to iterate through transactions, storing n-1 of them in tmp memory
		set_mem.w = PROGRAM(0);
			
		PROGRAM_X += 1;
		
		if (set_mem.w == 0) continue;

		set_mem.z = 0;

		//DO NOT use dynamic memory -> causes exponential decrease in performance
		//int* tmp_mem = (int*) malloc((set_mem.w) * sizeof(int));
		while (set_mem.z < set_mem.w)
		{
			++set_mem.z;
				
			uchar4 trans_mem;
			trans_mem.x = 0;
			trans_mem.y = PROGRAM(0);
			++PROGRAM_X;

			int ans = 0;
		
			while (trans_mem.x < trans_mem.y)
			{
				trans_mem.z = 0;
				trans_mem.w = PROGRAM(0);
				++PROGRAM_X;
				int mult = 1;

				trans_mem.x++;

				while (trans_mem.z < trans_mem.w)
				{
					trans_mem.z++;

					switch (PROGRAM(0))
					{
				//Currently unsupported
					/*case EXPR_READ_TMPMEM:
						mult *= tmp_mem[PROGRAM(1)];
						break;*/
					case EXPR_READ_SYSREG:
						//0 = length, 1 = value
						mult *= PROGRAM(1) == 0 ? (int)PROTO_LENGTH : value;
						break;
					case EXPR_READ_LOOKUP:
						mult *= lookup[PROGRAM(1)];
						break;
					}
					PROGRAM_X += 2;
				}
				ans += mult;
			}
				
			if (PROGRAM(0) == 0) // == EXPR_WRITE_SYSREG (1) - do for all threads
			{
				//Currently unsupported
				//tmp_mem[PROGRAM(1)] = ans;
			}
			else if (PROTO_ID == PROTO_CURR)// && PROGRAM(0) == EXPR_WRITE_TMPMEM (0)) - do for legitimate threads only
			{
				//can only write to length register
				PROTO_LENGTH = ans;
			}
				
			PROGRAM_X += 2;
		}
		//free(tmp_mem);
	}
}
#pragma endregion

#pragma region Cache


__device__ void GpfVm::CacheLoad(const  int* __restrict__ start_offset)
{
	//assume offset does not lie on integer lines
	
	int working = PROTOCOL_X + PROGRAM(0); //byte offset of current protocol + byte offset from start of protocol
	//FFFC = 11111111 11111100
		
	//int* start_offset = packet_data + (STREAM * PACKET_COUNT + packet_index & 0xFFFFFFFC) * _record_size  + CACHE_LANE; //group offset
	ALIGNMENT = working & 3;	//set byte alignment offset for extractions
	working = working >> 2; //get integer index in record

	//if (blockIdx.x == 0 && threadIdx.x < 4 && ITERATION < 4) 
	//{
	//	printf("Packet\n"); //%d :\n\tByte Offset = %d\n\tAlignment = %d\n\tIndex in Record = %d\n", packet_index, PROTOCOL_X + PROGRAM(0), ALIGNMENT, working);
	//}
	/*if (blockIdx.x == 0 && threadIdx.x < 4 && ITERATION < 4) 
	{
		printf("\nPacket %d :\n\tGroup Offset = %d\n\tRecord Size = %d\n\tCache Lane = %d\nWorking = %d\n", packet_index, group_offset,_record_size, CACHE_LANE, working);
	}*/
	
	++PROGRAM_X;
	
	cache[0] = __ldg(start_offset + __shfl(working, 0, 4));
	cache[1] = __ldg(start_offset + _record_size + __shfl(working, 1, 4));
	cache[2] = __ldg(start_offset + _record_size * 2 + __shfl(working, 2, 4));
	cache[3] = __ldg(start_offset + _record_size * 3 + __shfl(working, 3, 4));
		
	// 1,0,3,2
	working = __byte_perm(cache[transform.x], 0, 0x0123);
	cache[transform.x] = __shfl(working, static_cast<int>(transform.x), 4);

	// 3,2,1,0		
	working = __byte_perm(cache[transform.y], 0, 0x0123);
	cache[transform.y] = __shfl(working, static_cast<int>(transform.y), 4);

	// 2,3,0,1
	working = __byte_perm(cache[transform.z], 0, 0x0123);
	cache[transform.z] = __shfl(working, static_cast<int>(transform.z), 4);

	cache[CACHE_LANE] = __byte_perm(cache[CACHE_LANE], 0, 0x0123);
	/*if (blockIdx.x == 0 && threadIdx.x < 4 && ITERATION < 4) 
	{
		printf("\nPacket %d :\n\tCache = [ %x | %x | %x | %x ]\n", packet_index, cache[0], cache[1], cache[2], cache[3]);
	}*/
}
	
//extracts a field from the cache
//bit_offset - the bit offset of the field from the beginning of the segment, including any local shift
//bit_length - the bit length of the field
//returns - the field
__device__ unsigned int GpfVm::ExtractField()
{
	short2 field_offsets;
	field_offsets.x = (ALIGNMENT<<3) + PROGRAM(0); //bits to start of cache + bits from start of cache to field
	field_offsets.y = PROGRAM(1);
	PROGRAM_X += 2;

	//load cache integer, trimming leading bits
	unsigned int value = cache[field_offsets.x >> 5] & (0xFFFFFFFF >> (field_offsets.x & 31));
	
	//if field contained in one int, trim any trailing bits
	if ((field_offsets.x & 31) + field_offsets.y < 33) {
		value = value >> (32 - (field_offsets.x & 31) - field_offsets.y);
	}
	//else merge with next chunk
	else 
	{
		field_offsets.y = (field_offsets.x + field_offsets.y) & 31; 
		value = (value << field_offsets.y) + (cache[1 + (field_offsets.x >> 5)] >> (32 - field_offsets.y)); //shift off end bits
	}
	return value;
}


#pragma endregion


#pragma endregion


__global__ void GpfProcessor(int stream_no)
{
	GpfVm vm(stream_no);
	vm.Gather();
	vm.Filter();
}

#pragma region Host

__host__ void CheckError(const char* string)
{

#ifdef GPF_DEBUG
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("%s : %s\n\n", string, hipGetErrorString(error));
		getchar();
		exit(1);
	}
#endif

}

void ConstantSetup(FilterOptions options)
{
	int tmp;
	/*printf("\nFiltermem host: %d %d %d %d %d %d %d %d %d %d %d %d %d %d %d %d\n\n",
			options.filter_program[0], options.filter_program[1], options.filter_program[2], options.filter_program[3], 
			options.filter_program[4], options.filter_program[5], options.filter_program[6], options.filter_program[7],
			options.filter_program[8], options.filter_program[9], options.filter_program[10], options.filter_program[11], 
			options.filter_program[12], options.filter_program[13], options.filter_program[14], options.filter_program[15]);*/
	
	//unsigned char * dummy = (unsigned char*)malloc(16 + 20);
	//memset(dummy, 0, 3);
	//memcpy(dummy + 3, options.filter_program, 16);
	hipMemcpyToSymbol(HIP_SYMBOL(fltr_program), /*dummy*/ options.filter_program, options.filter_program_size);
	hipMemcpyToSymbol(HIP_SYMBOL(rule_program), options.rule_program, options.rule_program_size);
	hipMemcpyToSymbol(HIP_SYMBOL(lookup), options.lookup_memory, options.lookup_memory_size);
	
	hipMemcpyToSymbol(HIP_SYMBOL(_packet_count), (void*) &options.packets_per_buffer, sizeof(int));

	tmp = options.record_length / 4 + (options.record_length % 4 == 0 ? 0 : 1 );
	hipMemcpyToSymbol(HIP_SYMBOL(_record_size), (void*) &tmp, sizeof(int));

	hipMemcpyToSymbol(HIP_SYMBOL(_start_offset), (void*) &options.record_start, sizeof(int));
		
	hipMemcpyToSymbol(HIP_SYMBOL(_rules_per_packet), (void*) &options.rules_per_packet, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(_bool_result_count), (void*) &options.filters_per_packet, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(_int_result_count), (void*) &options.integers_per_packet, sizeof(int));


	hipMemcpyToSymbol(HIP_SYMBOL(_layer_count), (void*) &options.layer_count, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(_root_proto), (void*) &options.root_protocol, sizeof(int));
	
	tmp = options.rules_per_packet * options.packets_per_buffer / 32; //in ints
	hipMemcpyToSymbol(HIP_SYMBOL(_working_per_stream), (void*) &tmp, sizeof(int));

	tmp = options.rules_per_packet * GPF_THREADS; //in ints
	hipMemcpyToSymbol(HIP_SYMBOL(_working_per_block), (void*) &tmp, sizeof(int));

	tmp = options.rules_per_packet * 32; //in ints
	hipMemcpyToSymbol(HIP_SYMBOL(_working_per_warp), (void*) &tmp, sizeof(int));

	tmp = options.packets_per_buffer / 32;
	hipMemcpyToSymbol(HIP_SYMBOL(_filter_mem_size), (void*) &tmp, sizeof(int));		//size of filter mem per result in ints for a single result
	hipMemcpyToSymbol(HIP_SYMBOL(_integer_mem_size), (void*) &options.packets_per_buffer, sizeof(int));		//size of integer mem per result in ints for a single integer

	tmp = options.filter_memory_size() >> 2; 
	hipMemcpyToSymbol(HIP_SYMBOL(_filter_per_stream), (void*) &tmp, sizeof(int));
	tmp = options.integer_memory_size() >> 2; 
	hipMemcpyToSymbol(HIP_SYMBOL(_integer_per_stream), (void*) &tmp, sizeof(int));

}

void BeginProcess(void * args)
{
	int * packet_data_dev;
	int * filter_mem_dev;
	int * integer_mem_dev;
	int * working_mem_dev;
	
	ProcessArgs * proc = static_cast<ProcessArgs*>(args);
	FilterOptions options = proc->options;
	
	hipSetDevice(options.gpu_index);
	CheckError("Error setting device.");

	int device;
	hipGetDevice(&device);
	printf("\n\nDevice: %d\n\n", device);
	//connect to the cuda buffer object
	zmq::socket_t buffer(*proc->context, ZMQ_PAIR);
	buffer.connect("inproc://gpfbuffer");

	zmq::socket_t empty(*proc->context, ZMQ_PAIR);
	empty.connect("inproc://gpfbuffer_return");

	
	for (int k = 0; k < options.total_stream_buffers * options.streams; k++)
	{
		char* tmp;
		hipHostAlloc((void**) &tmp, options.packet_buffer_size(), hipHostMallocWriteCombined);
		empty.send(&tmp, sizeof(char*));
	}

	CudaBufferPointer ptr;

	//create an output buffer for vm results
	OutputBuffer output(*proc);

	
	//malloc device memory
	hipMalloc((void**) &packet_data_dev, options.packet_buffer_size() * options.streams);
	hipMalloc((void**) &filter_mem_dev, options.filter_memory_size() * options.streams);
	hipMalloc((void**) &integer_mem_dev, options.integer_memory_size() * options.streams);
	hipMalloc((void**) &working_mem_dev, options.working_memory_size() * options.streams);
	
	/*int * test = (int *)malloc(options.filter_memory_size() * options.streams);
	unsigned char val[2];
	val[0] = 0x0F;
	val[1] = 0xCC;

	for (int k = 0; k < options.streams;k++)
	{
		for (int j = 0; j < options.filters_per_packet;j++)
		{
			int count = options.packets_per_buffer/8;
			int offset = k * options.filter_memory_size() + j * count;
			memset((char*)test + offset, val[j], count);
		}
	}
	hipMemcpy(filter_mem_dev, test, options.filter_memory_size() * options.streams, hipMemcpyHostToDevice);*/
	
	CheckError("Error allocating device memory.");

	//copy pointers to constant memory
	hipMemcpyToSymbol(HIP_SYMBOL(packet_data), &packet_data_dev, sizeof(int *));		
	hipMemcpyToSymbol(HIP_SYMBOL(filter_mem), &filter_mem_dev, sizeof(int *));	
	hipMemcpyToSymbol(HIP_SYMBOL(integer_mem), &integer_mem_dev, sizeof(int *));	
	hipMemcpyToSymbol(HIP_SYMBOL(working_mem), &working_mem_dev, sizeof(int *));	

	CheckError("Error copying device memory pointers.");

	ConstantSetup(options);
	
	CheckError("Error during constant setup.");

	hipDeviceSetCacheConfig(hipFuncCachePreferL1);
	//hipDeviceSetLimit(hipLimitMallocHeapSize, 128 * 1024 * 1024);
	hipDeviceSetLimit(hipLimitPrintfFifoSize, 64 * 1024 * 1024);
	
	CheckError("Error during device heap setup.");

	_int64 packet_count = 0;

	int filter_size = options.filter_memory_size();
	int integer_size = options.integer_memory_size();
	
	//create streams
    hipStream_t *streams = (hipStream_t*) malloc(options.streams * sizeof(hipStream_t));
    
	for(int k = 0; k < options.streams; k++) 
	{
        hipStreamCreateWithFlags(&(streams[k]), hipStreamNonBlocking);
    }
	CheckError("Error creating streams.");
	
	int** filter_results = (int**) malloc(options.streams * sizeof(int*));
	int** integer_results = (int**) malloc(options.streams * sizeof(int*));

	size_t* stream_size = (size_t*)malloc(options.streams * sizeof(size_t));
	int* packets = (int*)malloc(options.streams * sizeof(int));
	char ** packet_ptrs = (char**)malloc(options.streams * sizeof(char*));

//#ifdef PERFORMANCE_TESTING
//	hipEvent_t start;
//	hipEvent_t* events = (hipEvent_t*)malloc(sizeof(hipEvent_t) * options.streams * 4);
//	GpfTimer timer("H:\\testing.csv", &options);
//
//	for (int k = 0; k < options.streams * 4; k++)
//	{
//		hipEventCreate(&events[k]);
//	}
//
//	hipEventCreate(&start);
//	hipEventRecord(start, 0);
//
//#endif

	do
	{
		
		int usedStreams = 0;
		for (int k = 0; k < options.streams; k++)
		{
			buffer.recv(&ptr, sizeof(CudaBufferPointer));	//get next full write-combined buffer

			if (ptr.size == 0) break; //skip if stream is empty
			
			usedStreams++;
			stream_size[k] = ptr.size;
			packet_ptrs[k] = ptr.buffer;
			packets[k] = ((int)ptr.size) / options.record_length;

			packet_count += packets[k];
		
			//if nonstandard packet count, update constant memory (final iteration)
			if (packets[k] != options.packets_per_buffer)	
			{
				hipMemcpyToSymbolAsync(HIP_SYMBOL(_packet_count), &packets[k], sizeof(int), 0, hipMemcpyHostToDevice, 0); //copy in default stream so other kernels finish first
				//CheckError("Error resetting const memory.");
			}
			
			char * dst = ((char*) packet_data_dev) + k * options.packet_buffer_size();
			
//#ifdef PERFORMANCE_TESTING
//			//issue
//			hipEventRecord(events[k * 4], streams[k]);
//#endif
			//copy records to the device - async
			hipMemcpyAsync(dst, ptr.buffer, ptr.size, hipMemcpyHostToDevice, streams[k]);
			//CheckError("Error streaming packet data.");
		
			
			int blocks = static_cast<int>(ceil(static_cast<double>(packets[k]) / PACKETS_PER_BLOCK));
				
//#ifdef PERFORMANCE_TESTING
//			//load
//			hipEventRecord(events[k * 4 + 1], streams[k]);
//#endif
			
			//process stream contents - async
			GpfProcessor<<<blocks, GPF_THREADS, 0, streams[k]>>>(k);
			//CheckError("Error in vm.");
						
			hipStreamSynchronize(streams[k]); //shouldnt be necessary? but seems to prevent corruption on smaller captures
//#ifdef PERFORMANCE_TESTING
//			//process
//			hipEventRecord(events[k * 4 + 2], streams[k]);
//#endif
			int* tmp;
			if (options.filters_per_packet > 0)  
			{	
				tmp = output.GetFilterBuffer();

				/*hipHostAlloc((void**) &tmp, filter_size, hipHostMallocDefault); 
				CheckError("Error allocating host filter output buffers.");*/
				
				hipMemcpyAsync(tmp, (char*)(filter_mem_dev) + k * filter_size, filter_size, hipMemcpyDeviceToHost, streams[k]);
				//CheckError("Error copying host filter output buffers.");


				filter_results[k] = tmp;
			}
			if (options.integers_per_packet > 0) 
			{
				tmp = output.GetFieldBuffer();
				/*hipHostAlloc((void**) &tmp, integer_size, hipHostMallocDefault); 
				CheckError("Error allocating host integer output buffers.");*/

				hipMemcpyAsync(tmp, (char*)(integer_mem_dev) + k * integer_size, integer_size, hipMemcpyDeviceToHost, streams[k]);
				//CheckError("Error copying host integer output buffers.");

				integer_results[k] = tmp;
			}
										
//#ifdef PERFORMANCE_TESTING
//			//return
//			hipEventRecord(events[k * 4 + 3], streams[k]);
//#endif
			if (!ptr.more) break;
		}			
		
		for (int k = 0; k < usedStreams; k++)
		{
			hipStreamSynchronize(streams[k]);
			//CheckError("Error synchronizing stream.");
			//free the packet buffer as it is no longer needed
			char* tmp = packet_ptrs[k];
			empty.send(&tmp, sizeof(char*));

			if (options.filters_per_packet > 0) output.CopyFilterResults(filter_results[k], filter_size, packets[k]);
			//CheckError("Error copying to host filter output buffers.");

			if (options.integers_per_packet > 0) output.CopyIntegerResults(integer_results[k], integer_size, packets[k]);
			//CheckError("Error copying to host integer output buffers.");
													
//#ifdef PERFORMANCE_TESTING
//			//return
//			float issue;
//			float load;
//			float process;
//			float conclude;
//
//			hipEventElapsedTime(&issue, start, events[k * 4]);
//			hipEventElapsedTime(&load, events[k * 4], events[k * 4 + 1]);
//			hipEventElapsedTime(&process, events[k * 4 + 1], events[k * 4 + 2]);
//			hipEventElapsedTime(&conclude, events[k * 4 + 2], events[k * 4 + 3]);
//
//			timer.Record(packets[k], k, issue, load, process, conclude);
//#endif

		}

	} while (ptr.more);
	//complete

	output.Finished(packet_count);

	//malloc device memory
	hipFree((void*)packet_data_dev);
	hipFree((void*)filter_mem_dev);
	hipFree((void*)integer_mem_dev);

	
	for(int k = 0; k < options.streams; k++) 
	{
        hipStreamDestroy(streams[k]);
    }

	free(stream_size);
	free(packets);
	free(packet_ptrs);

	free(streams);

	empty.close();
	buffer.close();
}

//launches a new vm thread with the prescribed filter options
void GpfVmLauncher(zmq::context_t * zmq_context, FilterOptions filter_options)
{
	//create argument for vm thread
	ProcessArgs * args = static_cast<ProcessArgs*>(malloc(sizeof(ProcessArgs)));
	args->context = zmq_context;
	args->options = filter_options;

	//issue thread
	tthread::thread* proc = new tthread::thread(BeginProcess, (void*) args);
}


#pragma endregion


//GpfTimer::GpfTimer(char* output_file, FilterOptions* options)
//{
//	streams = options->streams;
//	gpu = options->gpu_index;
//	filename = output_file;
//	bool exists;
//
//	if (FILE *file = fopen(filename, "r")) {
//        fclose(file);
//        exists = true;
//    } else exists = false;
//    
//	fopen_s(&file, filename, "a");
//
//	if (exists)	fprintf(file, "Packet Count,Stream ID,Issue Time,Host To Device,Classification,Device To Host\n");
//	
//	fprintf(file, "\nStream Count,%d\nGPU Index,%d\n",streams,gpu);
//}
//GpfTimer::~GpfTimer()
//{
//	fflush(file);
//	fclose(file);
//	//free(filename);
//}
//void GpfTimer::Record(int packet_count, int stream_id, float issueTime, float packetCopy, float packetProcess, float resultCopy)
//{
//	fprintf(file, "%d,%d,%f,%f,%f,%f\n", packet_count, stream_id, issueTime, packetCopy, packetProcess, resultCopy);
//}