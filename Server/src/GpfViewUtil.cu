#include "hip/hip_runtime.h"

#include "zmq.hpp"
#include "thrust\device_vector.h"
#include "thrust\reduce.h"
#include "thrust\sort.h"
#include <thrust\fill.h>
#include <thrust\execution_policy.h>

#include "GpfViewUtil.cuh"
#include "Distiller.h"


__global__ void CountGlobal(int* filterSegment, int segmentSize, unsigned long long int* resultPtr)
{
	int count = (blockDim.x * blockIdx.x + threadIdx.x < segmentSize ? __popc(filterSegment[blockDim.x * blockIdx.x + threadIdx.x]) : 0);
	
	count += __shfl_xor(count, 16);
	count += __shfl_xor(count, 8);
	count += __shfl_xor(count, 4);
	count += __shfl_xor(count, 2);
	count += __shfl_xor(count, 1);

	if ((threadIdx.x & 31) == 0) atomicAdd(resultPtr, count);
}


#define VIEW_UTIL_COUNT 0
#define VIEW_UTIL_DISTILL 1
#define VIEW_UTIL_EXIT 2

void CountProcess(zmq::socket_t& socket)
{
	unsigned long long int*  count_ptrs_dev;
	int * dev_ptr;
	int total_segments;
	int max_seg_size;
	int device;
	/*int streams;
	int stream_segments;*/
		
	socket.recv(&device, sizeof(int));
	hipSetDevice(device);
	hipDeviceReset();

	socket.recv(&total_segments, sizeof(int));

	//if (total_segments > 128)
	//{
	//	streams = 128;
	//	stream_segments = 1 + total_segments / 128;
	//}
	//else 
	//{
	//	streams = total_segments;
	//	stream_segments = 1;
	//}

	hipStream_t * streams = (hipStream_t *)malloc(total_segments * sizeof(hipStream_t));

	for (int k = 0; k < total_segments; k++) hipStreamCreate(&(streams[k]));
	
	int count_mem_size = total_segments * sizeof(unsigned long long int);
	hipMalloc((void **) &count_ptrs_dev, count_mem_size);
	
	socket.recv(&max_seg_size, sizeof(int));
	hipMalloc((void**) &dev_ptr, max_seg_size * total_segments);

	hipMemsetAsync(count_ptrs_dev, 0, count_mem_size, 0);

	int max_size_ints = max_seg_size / 4;

	for (int k = 0; k < total_segments; k++)
	{
		zmq::message_t msg;
		socket.recv(&msg);

		int * ptr = dev_ptr + k * max_size_ints;
		hipMemcpyAsync(ptr, msg.data(), msg.size(), hipMemcpyHostToDevice, streams[k]);

		int msg_size_int = msg.size() / 4;
		int blocks = msg_size_int / 128 + (msg_size_int % 128 == 0 ? 0 : 1);

		CountGlobal<<<blocks, 128, 0, streams[k]>>>(ptr, msg_size_int, count_ptrs_dev + k);
	}
	
	int * count_ptrs_host = (int *) malloc(count_mem_size);
	
	hipDeviceSynchronize();
	
	hipMemcpy(count_ptrs_host, count_ptrs_dev, count_mem_size, hipMemcpyDeviceToHost);

	socket.send(count_ptrs_host, count_mem_size);
	
	for (int k = 0; k < total_segments; k++) hipStreamDestroy(streams[k]);

	free(streams);
	hipFree(dev_ptr);
	hipFree(count_ptrs_dev);
	free(count_ptrs_host);
}


//
//void ReduceProcess(zmq::socket_t& socket)
//{
//	int total_segments;
//	int segment_size;
//
//	socket.recv(&total_segments, sizeof(int));
//	socket.recv(&segment_size, sizeof(int));
//	
//	int* keys_in_raw;
//	int* values_in_raw;
//	int* keys_out_raw;
//	int* values_out_raw;
//
//	int* values_in_host;
//	int* keys_out_host;
//	int* values_out_host;
//
//	hipHostAlloc((void**)&values_in_host, segment_size * sizeof(int), hipHostMallocWriteCombined);
//	hipHostMalloc((void**)&keys_out_host, segment_size * sizeof(int));
//	hipHostMalloc((void**)&values_out_host, segment_size * sizeof(int));
//
//	hipMalloc((void**)&keys_in_raw, segment_size * sizeof(int));
//	hipMalloc((void**)&values_in_raw, segment_size * sizeof(int));
//	hipMalloc((void**)&keys_out_raw, segment_size * sizeof(int));
//	hipMalloc((void**)&values_out_raw, segment_size * sizeof(int));
//
//	for (int k = 0; k < segment_size; k++)
//	{
//		values_in_host[k] = 1;
//	}
//	
//	hipMemcpy(values_in_raw, values_in_host, segment_size * sizeof(int), hipMemcpyHostToDevice);
//	
//	/*hipStream_t * streams = (hipStream_t *)malloc(total_segments * sizeof(hipStream_t));
//	for (int k = 0; k < total_segments; k++) hipStreamCreate(&(streams[k]));*/
//	
//	for (int k = 0; k < total_segments; k++)
//	{
//		zmq::message_t msg;
//		socket.recv(&msg);
//
//		int * data = (int*)msg.data();
//		int count = msg.size() / sizeof(int);
//
//		hipMemcpy(keys_in_raw, msg.data(), msg.size(), hipMemcpyHostToDevice);
//		
//		if (k + 1 == total_segments)
//		{
//			hipMemset(keys_out_raw, 0, msg.size());
//			hipMemset(values_out_raw, 0, msg.size());
//		}
//
//		thrust::device_ptr<int> keys_in(keys_in_raw);
//		thrust::device_ptr<int> values_in(values_in_raw);
//		thrust::device_ptr<int> keys_out(keys_out_raw);
//		thrust::device_ptr<int> values_out(values_out_raw);
//
//		thrust::sort(keys_in, keys_in + count);
//		thrust::pair<thrust::device_ptr<int>,thrust::device_ptr<int>> result = thrust::reduce_by_key(keys_in, keys_in + count, values_in, keys_out, values_out);
//		
//		thrust::copy(keys_out, result.first, keys_out_host);
//		thrust::copy(values_out, result.second, values_out_host);
//		
//		socket.send(keys_out_host, segment_size * sizeof(int));
//		socket.send(values_out_host, segment_size * sizeof(int));
//	}
//		
//	/*for (int k = 0; k < total_segments; k++) hipStreamDestroy(streams[k]);
//	free(streams);*/
//}


void GpfViewUtil(zmq::context_t& context, int port)
{
	zmq::socket_t socket(context, ZMQ_PAIR);
	char * str = (char*)malloc(25);
	sprintf(str, "tcp://*:%d\0",port);
	socket.bind(str);
	bool running = true;
	
	while(running)
	{
		int command;
		socket.recv(&command, sizeof(int));

		switch (command)
		{
		case VIEW_UTIL_COUNT:
			CountProcess(socket);	
			break;
		case VIEW_UTIL_DISTILL:
			Distiller(context, socket);
			break;
		case VIEW_UTIL_EXIT:
			running = false;
			break;
		default: 
			break;
		}

	}
	socket.close();
	free(str);
}